#include "hip/hip_runtime.h"
/*
 * Part of LAKE: Towards a Machine Learning-Assisted Kernel with LAKE
 * Copyright (C) 2022-2024 Henrique Fingler
 * Copyright (C) 2022-2024 Isha Tarte
 * 
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>
#include "test_weights.h"
#define LEN_INPUT 31
#define LEN_LAYER_0 256
#define LEN_LAYER_0_HALF 128
#define LEN_LAYER_1 2

__global__ void prediction_mid_layer_batch(long *weight_0_T_ent, long *bias_0_ent, long *input_vec_i, long *mid_res_i) { 
	int j, offset;

	int threadId = threadIdx.x;
    int stride = blockDim.x;
	int input_ind = blockIdx.x*LEN_INPUT;
	int blockId = blockIdx.x;
	for (j = threadId, offset=threadId*LEN_INPUT; j < LEN_LAYER_0; j+=stride, offset+=LEN_INPUT*stride) {
		int update_index = blockId*stride + j;
        mid_res_i[update_index] = 0;
		//loop unroll
		mid_res_i[update_index] =  mid_res_i[update_index] + input_vec_i[input_ind + 0] * weight_0_T_ent[offset+0]
		+ input_vec_i[input_ind + 1] * weight_0_T_ent[offset+1]
		+ input_vec_i[input_ind + 2] * weight_0_T_ent[offset+2]
		+ input_vec_i[input_ind + 3] * weight_0_T_ent[offset+3]
		+ input_vec_i[input_ind + 4] * weight_0_T_ent[offset+4]
		+ input_vec_i[input_ind + 5] * weight_0_T_ent[offset+5]
		+ input_vec_i[input_ind + 6] * weight_0_T_ent[offset+6]
		+ input_vec_i[input_ind + 7] * weight_0_T_ent[offset+7]
		+ input_vec_i[input_ind + 8] * weight_0_T_ent[offset+8]
		+ input_vec_i[input_ind + 9] * weight_0_T_ent[offset+9]
		+ input_vec_i[input_ind + 10] * weight_0_T_ent[offset+10]
		+ input_vec_i[input_ind + 11] * weight_0_T_ent[offset+11]
		+ input_vec_i[input_ind + 12] * weight_0_T_ent[offset+12]
		+ input_vec_i[input_ind + 13] * weight_0_T_ent[offset+13]
		+ input_vec_i[input_ind + 14] * weight_0_T_ent[offset+14]
		+ input_vec_i[input_ind + 15] * weight_0_T_ent[offset+15]
		+ input_vec_i[input_ind + 16] * weight_0_T_ent[offset+16]
		+ input_vec_i[input_ind + 17] * weight_0_T_ent[offset+17]
		+ input_vec_i[input_ind+ 18] * weight_0_T_ent[offset+18]
		+ input_vec_i[input_ind + 19] * weight_0_T_ent[offset+19]
		+ input_vec_i[input_ind + 20] * weight_0_T_ent[offset+20]
		+ input_vec_i[input_ind + 21] * weight_0_T_ent[offset+21]
		+ input_vec_i[input_ind + 22] * weight_0_T_ent[offset+22]
		+ input_vec_i[input_ind + 23] * weight_0_T_ent[offset+23]
		+ input_vec_i[input_ind + 24] * weight_0_T_ent[offset+24]
		+ input_vec_i[input_ind + 25] * weight_0_T_ent[offset+25]
		+ input_vec_i[input_ind + 26] * weight_0_T_ent[offset+26]
		+ input_vec_i[input_ind + 27] * weight_0_T_ent[offset+27]
		+ input_vec_i[input_ind + 28] * weight_0_T_ent[offset+28]
		+ input_vec_i[input_ind + 29] * weight_0_T_ent[offset+29]
		+ input_vec_i[input_ind + 30] * weight_0_T_ent[offset+30];

        // apply bias
        mid_res_i[update_index] += bias_0_ent[threadId];
        // relu
        if (mid_res_i[update_index] < 0) {
            mid_res_i[update_index] = 0;
        }		
    }
}

__global__ void prediction_mid_layer_1_batch(long *weight_M_1, long *bias_M_1, long *mid_res_i, long *mid_res_1_i) { 
	int j, offset, k;

	int threadId = threadIdx.x;
    int stride = blockDim.x;
	int input_ind = blockIdx.x*256;
	int blockId = blockIdx.x;
	for (j = threadId, offset=threadId*256; j < LEN_LAYER_0; j+=stride, offset+=256*stride) {
		int update_index = blockId*stride + j;
        mid_res_1_i[update_index] = 0;
		//loop unroll
		for(k = 0; k < 256; k++) {
			mid_res_1_i[update_index] += weight_M_1[offset + k] * mid_res_i[input_ind + k];
		}

        // // apply bias
        mid_res_1_i[update_index] += bias_M_1[threadId];
        // relu
        if (mid_res_1_i[update_index] < 0) {
            mid_res_1_i[update_index] = 0;
        }		
    }
}

__global__ void prediction_mid_layer_2_batch(long *weight_M_2, long *bias_M_2, long *mid_res_1_i, long *mid_res_2_i) { 
	int j, offset, k;

	int threadId = threadIdx.x;
    int stride = blockDim.x;
	int input_ind = blockIdx.x*256;
	int blockId = blockIdx.x;
	for (j = threadId, offset=threadId*256; j < LEN_LAYER_0; j+=stride, offset+=256*stride) {
		int update_index = blockId*stride + j;
        mid_res_2_i[update_index] = 0;
		//loop unroll
		for(k = 0; k < 256; k++) {
			mid_res_2_i[update_index] += weight_M_2[offset + k] * mid_res_1_i[input_ind + k];
		}

        // apply bias
        mid_res_2_i[update_index] += bias_M_2[threadId];
        // relu
        if (mid_res_2_i[update_index] < 0) {
            mid_res_2_i[update_index] = 0;
        }		
    }
}

__global__ void prediction_final_layer_batch(long *weight_1_T_ent, long *bias_1_ent, long *mid_res_i, long *dd_final_res_i) {
	int index = blockIdx.x;
	int threadId = threadIdx.x;
	int dim = blockDim.x;
	int k;
	int update_index = index*dim + threadId;
	if (threadId < 32) {
		dd_final_res_i[update_index] = 0;
		for(k = threadId; k<LEN_LAYER_0; k = k + 32) {
			dd_final_res_i[update_index] =  dd_final_res_i[update_index] + mid_res_i[index*LEN_LAYER_0 + k] * weight_1_T_ent[k];
		}
	} else {
		dd_final_res_i[update_index] = 0;
		for(k = threadId - 32; k<LEN_LAYER_0; k = k + 32) {
			dd_final_res_i[update_index] =  dd_final_res_i[update_index] + mid_res_i[index*LEN_LAYER_0 + k] * weight_1_T_ent[k+256];
		}
	}
	__syncthreads();
	if (threadId == 0) {
		update_index = index*dim;
		for(int i = 1; i < 32; i++) {
			dd_final_res_i[update_index] = dd_final_res_i[update_index] + dd_final_res_i[update_index + i];
		}
		dd_final_res_i[update_index] =  dd_final_res_i[update_index] + bias_1_ent[0];
	}
	if(threadId == 32) {
		update_index = index*dim + 32;
		for(int i = 1; i < 32; i++) {
			dd_final_res_i[update_index] = dd_final_res_i[update_index] + dd_final_res_i[update_index + i];
		} 
		dd_final_res_i[update_index] =  dd_final_res_i[update_index] + bias_1_ent[1];
	}
}

// static long *weight_0_T_ent, * bias_0_ent, *weight_1_T_ent, * bias_1_ent; 
// static long input_vec_i[31] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,0,0,0,0,9,0,0,0,9,0,0,0,9};
// static long *parallel_input;
// static long *final_res_i;
// static long *d_weight_0_T_ent, *d_weight_1_T_ent, *d_bias_0_ent, *d_bias_1_ent, *d_input_vec_i, *d_mid_res_i, *d_final_res_i;


// void copy_inputs_batch(int batch_size) {
// 	hipMemcpy(d_input_vec_i, parallel_input, sizeof(long) * 31 * batch_size, hipMemcpyHostToDevice);
// }

// void infer_batch(int batch_size) {
// 	prediction_mid_layer_batch<<<batch_size,256>>>(d_weight_0_T_ent, d_bias_0_ent, d_input_vec_i, d_mid_res_i);
// 	prediction_final_layer_batch<<<batch_size, 64>>>(d_weight_1_T_ent, d_bias_1_ent, d_mid_res_i, d_final_res_i);
// }

// bool get_result_batch(int batch_size) {
// 	hipMemcpy(final_res_i, d_final_res_i, sizeof(long) * 64 * batch_size, hipMemcpyDeviceToHost);
	
// 	bool res[batch_size];
// 	for(int i = 0; i < batch_size; i++) {
// 		// printf("\n %ld", final_res_i[i*64]);
// 		// printf("\n %ld", final_res_i[i*64 + 32]);
// 		res[i] = final_res_i[i*64]>=(final_res_i[i *64 + 32])? false: true;
// 	}

// 	return res[0];
// }

// void setup_batch(int batch_size) {
// 	final_res_i = new long[batch_size*64];
// 	parallel_input = new long[batch_size*31];

// 	for(int b = 0 ; b < batch_size; b++) {
// 		for(int j = 0; j < 31; j++)
// 			parallel_input[ b*31 + j ] = input_vec_i[j];
// 	}
	
// 	weight_0_T_ent = &weight_i_0_T[0][0];
// 	weight_1_T_ent = &weight_i_1[0][0];
// 	bias_0_ent = bias_i_0;
// 	bias_1_ent = bias_i_1;

// 	hipMalloc((void**)&d_weight_0_T_ent, sizeof(long) * 256*31);
// 	hipMalloc((void**)&d_weight_1_T_ent, sizeof(long) * 256*2);
// 	hipMalloc((void**)&d_bias_0_ent, sizeof(long) * 256);
// 	hipMalloc((void**)&d_bias_1_ent, sizeof(long) *2);

// 	hipMalloc((void**)&d_mid_res_i, sizeof(long) *LEN_LAYER_0 * batch_size);
// 	hipMalloc((void**)&d_final_res_i, sizeof(long) *LEN_LAYER_1 * batch_size *32);

// 	hipMalloc((void**)&d_input_vec_i, sizeof(long) *LEN_INPUT * batch_size);
// 	hipMemcpy(d_weight_0_T_ent, weight_0_T_ent, sizeof(long) * 256*31, hipMemcpyHostToDevice);
// 	hipMemcpy(d_weight_1_T_ent, weight_1_T_ent, sizeof(long) * 256*2, hipMemcpyHostToDevice);
// 	hipMemcpy(d_bias_0_ent, bias_0_ent, sizeof(long) * 256, hipMemcpyHostToDevice);
// 	hipMemcpy(d_bias_1_ent, bias_1_ent, sizeof(long) * 2, hipMemcpyHostToDevice);
// 	printf("done %d\n", batch_size);
// }

// void clean_batch() {
// 	hipFree(d_input_vec_i);
// 	hipFree(d_weight_0_T_ent);
// 	hipFree(d_weight_1_T_ent);
// 	hipFree(d_bias_0_ent);
// 	hipFree(d_bias_1_ent);
// 	hipFree(d_mid_res_i);
// 	hipFree(d_final_res_i);
// 	delete final_res_i;
// 	delete parallel_input;
// }
